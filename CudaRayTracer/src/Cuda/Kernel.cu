#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>

#include "Hittables/Hittable.cuh"
#include "Hittables/HittableList.cuh"
#include "Hittables/Material.cuh"
#include "Utils/SharedStructs.h"

// convert floating point rgb color to 8-bit integer
__device__ inline int RgbToInt(float r, float g, float b)
{
    r = Clamp(r, 0.0f, 255.0f);
    g = Clamp(g, 0.0f, 255.0f);
    b = Clamp(b, 0.0f, 255.0f);
    float a = 255.0f;
    return (int(a) << 24) | (int(b) << 16) | (int(g) << 8) | int(r);
}

// the reverse
__device__ inline Vec3 IntToRgb(int val)
{
    float r = val % 256;
    float g = (val % (256 * 256)) / 256;
    float b = val / (256 * 256);
    return Vec3(r, g, b);
}

// __device__ bool Hit(const Ray& r, float t_min, float t_max, HitRecord& rec, Hittable** world, unsigned int
// world_size)
// {
//     HitRecord temp_rec;
//     bool hit_anything = false;
//     float closest_so_far = t_max;

//     for (int i = 0; i < world_size; i++) {
//         if (world[i]->Hit(r, t_min, closest_so_far, temp_rec)) {
//             hit_anything = true;
//             closest_so_far = temp_rec.t;
//             rec = temp_rec;
//         }
//     }

//     return hit_anything;
// }

__device__ inline Vec3 color(const Ray& r, Hittable* world, int max_depth, hiprandState* local_rand_state)
{
    Ray cur_ray = r;
    Vec3 cur_attenuation = Vec3(1.0f, 1.0f, 1.0f);
    Vec3 background = Vec3(0.0f, 0.0f, 0.0f);

    HitRecord rec;

    for (int i = 0; i < max_depth; i++) {
        if (!world->Object->bvh_node->Hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            Vec3 unit_direction = UnitVector(cur_ray.Direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            Vec3 c = (1.0f - t) * Vec3(1.0f, 1.0f, 1.0f) + t * Vec3(0.5f, 0.7f, 1.0f);
            return cur_attenuation * c;
        }
        else {
            Vec3 emitted = Vec3(0.0f, 0.0f, 0.0f);
            Ray scattered;
            Vec3 attenuation;

            switch (rec.mat_ptr->type) {
            case MaterialType::LAMBERTIAN:
                if (!rec.mat_ptr->Object->lambertian->Scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                    return emitted * cur_attenuation;
                }
                break;
            case MaterialType::METAL:
                if (!rec.mat_ptr->Object->metal->Scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                    return emitted * cur_attenuation;
                }
                break;
            case MaterialType::DIELECTRIC:
                if (!rec.mat_ptr->Object->dielectric->Scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                    return emitted * cur_attenuation;
                }
                break;
            case MaterialType::DIFFUSELIGHT:
                emitted = rec.mat_ptr->Object->diffuse_light->Emitted(rec.u, rec.v, rec.p);
                return emitted * cur_attenuation;
            default:
                return background;
            }

            cur_attenuation = attenuation * cur_attenuation;
            cur_ray = scattered;
        }
    }

    return background; // exceeded recursion
}

__device__ inline void GetXYZCoords(int& x, int& y, int& z)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int bt = blockDim.z;
    x = blockIdx.x * bw + tx;
    y = blockIdx.y * bh + ty;
    z = blockIdx.z + bt * tz;
}

#define THREADS_PER_BLOCK 256
#if __CUDA_ARCH__ >= 200
#define MY_KERNEL_MAX_THREADS (4 * THREADS_PER_BLOCK)
#else
#define MY_KERNEL_MAX_THREADS THREADS_PER_BLOCK
#endif

__global__ __launch_bounds__(MY_KERNEL_MAX_THREADS) void Kernel(unsigned int* pos, unsigned int width,
                                                                unsigned int height,
                                                                const unsigned int samples_per_pixel,
                                                                const unsigned int max_depth, Hittable* world,
                                                                hiprandState* rand_state, InputStruct inputs)
{
    // extern __shared__ uchar4 sdata[];
    // Define shared memory for the rand_state array.
    // Each thread in the block will have one hiprandState element.
    // extern __shared__ hiprandState shared_rand_state[];

    int x, y, z;
    GetXYZCoords(x, y, z);

    if ((x >= width) || (y >= height))
        return;

    unsigned int pixel_index = (y * width + x);

    // Copy from global to shared memory
    // shared_rand_state[threadIdx.y * blockDim.x + threadIdx.x] = rand_state[pixel_index];
    hiprandState local_rand_state = rand_state[pixel_index];
    // Make sure all threads have finished copying
    // __syncthreads();
    // hiprandState local_rand_state = shared_rand_state[threadIdx.y * blockDim.x + threadIdx.x];

    Vec3 col = Vec3(0.0f, 0.0f, 0.0f);

    Vec3 origin = Vec3(inputs.origin_x, inputs.origin_y, inputs.origin_z);
    Vec3 forwardV = Vec3(inputs.orientation_x, inputs.orientation_y, inputs.orientation_z);
    Vec3 upV = Vec3(inputs.up_x, inputs.up_y, inputs.up_z);
    Vec3 rightV = Normalize(Cross(upV, forwardV));

    Vec3 center = Vec3(width / 2.0f, height / 2.0f, 0.0f);

    for (int s = 0; s < samples_per_pixel; s++) {
        // calculate uv coordinates
        float u = (float)((x - center.x()) + hiprand_uniform(&local_rand_state)) / (float)(width);
        float v = (float)((center.y() - y) + hiprand_uniform(&local_rand_state)) / (float)(width);
        Vec3 distFromCenter = (u * rightV) + (v * upV);
        Vec3 startPos = (inputs.near_plane * distFromCenter) + origin + (inputs.fov * forwardV);
        Vec3 secondPlanePos = (inputs.far_plane * distFromCenter) + ((1.0f / inputs.fov * 10.0f) * forwardV) + origin;
        Vec3 dirVector = Normalize(secondPlanePos - startPos);

        Ray r = Ray(startPos, dirVector);
        col = col + color(r, world, max_depth, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;

    col = col / (float)(samples_per_pixel);
    col.e[0] = 255.0f * sqrtf(col.x());
    col.e[1] = 255.0f * sqrtf(col.y());
    col.e[2] = 255.0f * sqrtf(col.z());

    // write output vertex
    pos[pixel_index] = RgbToInt(col.x(), col.y(), col.z());
}

__global__ void RandInit(hiprandState* rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
        hiprand_init(1984, 0, 0, rand_state);
}

__global__ void RenderInit(unsigned int window_width, unsigned int window_height, hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= window_width) || (j >= window_height))
        return;

    unsigned int pixel_index = (j * window_width + i);
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

// __global__ void CreateWorld(Sphere** d_list, Sphere** d_world, hiprandState* rand_state)
// {
//     if (threadIdx.x == 0 && blockIdx.x == 0) {
//         hiprandState local_rand_state = *rand_state;

//         int i = 0;

//         d_list[i++] = new Sphere(Vec3(0, -1000.0, -1), 1000,
//            new Material(Vec3(0.5, 0.5, 0.5), Mat::lambertian));
//         for (int a = -2; a < 2; a++) {
//            for (int b = -2; b < 2; b++) {
//                float choose_mat = RND;
//                Vec3 center = Vec3(a + RND, 0.2, b + RND);
//                if (choose_mat < 0.8f) {
//                    d_list[i++] = new Sphere(center, 0.2,
//                        new Material(Vec3(RND * RND, RND * RND, RND * RND), Mat::lambertian));
//                }
//                else if (choose_mat < 0.95f) {
//                    d_list[i++] = new Sphere(center, 0.2,
//                        new Material(Vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND,
//                        Mat::metal));
//                }
//                else {
//                    d_list[i++] = new Sphere(center, 0.2, new Material(1.5, Mat::dielectric));
//                }
//            }
//         }
//         d_list[i++] = new Sphere(Vec3(0, 1, 0), 1.0, new Material(1.5, Mat::dielectric));
//         d_list[i++] = new Sphere(Vec3(-4, 1, 0), 1.0, new Material(Vec3(0.4, 0.2, 0.1), Mat::lambertian));
//         d_list[i++] = new Sphere(Vec3(4, 1, 0), 1.0, new Material(Vec3(0.7, 0.6, 0.5), 0.0, Mat::metal));

//         // d_list[i++] = new Sphere(Vec3(0, -100.5, 0), 100, new Material(Vec3(0.8, 0.8, 0.0), Mat::lambertian));
//         // d_list[i++] = new Sphere(Vec3(0, 0, -1), 0.5, new Material(Vec3(0.1, 0.2, 0.5), Mat::lambertian));
//         // d_list[i++] = new Sphere(Vec3(1, 0, -1), 0.5, new Material(Vec3(0.8, 0.6, 0.2), 0.0, Mat::metal));
//         // d_list[i++] = new Sphere(Vec3(-1, 0, -1), 0.5, new Material(1.5f, Mat::dielectric));
//         // d_list[i++] = new Sphere(Vec3(-1, 0, -1), -0.45, new Material(1.5f, Mat::dielectric));

//         *rand_state = local_rand_state;
//         *d_world = new HittableList(d_list, i);
//     }
// }

// __global__ void FreeWorld(Sphere** d_list, Sphere** d_world, const unsigned int num_hittables) {
//     for (int i = 0; i < num_hittables; i++) {
//         delete ((Sphere*)d_list[i])->mat_ptr;
//         delete d_list[i];
//     }
//     delete* d_world;
// }

extern "C" void LaunchKernel(unsigned int* pos, unsigned int image_width, unsigned int image_height,
                             const unsigned int samples_per_pixel, const unsigned int max_depth, Hittable* world,
                             hiprandState* d_rand_state, InputStruct inputs)
{
    // Calculate grid size
    dim3 block(16, 16, 1);
    dim3 grid(image_width / block.x, image_height / block.y, 1);
    // Calculate the size of shared memory:
    // number of threads per block * size of each hiprandState element
    // size_t sbytes = block.x * block.y * sizeof(hiprandState);

    // HittableList* d_world;
    // hipMallocManaged((void**)&d_world, world->objects.size() * sizeof(HittableList));

    // for (int i = 0; i < world->objects.size(); i++) {
    //     d_world[i] = world->objects[i];
    // }

    Kernel<<<grid, block>>>(pos, image_width, image_height, samples_per_pixel, max_depth, world, d_rand_state, inputs);
    hipDeviceSynchronize();

    // hipFree(d_world);
}

extern "C" void LaunchRandInit(hiprandState* d_rand_state2)
{
    RandInit<<<1, 1>>>(d_rand_state2);
    hipDeviceSynchronize();
}

extern "C" void LaunchRenderInit(dim3 grid, dim3 block, unsigned int window_width, unsigned int window_height,
                                 hiprandState* d_rand_state)
{
    RenderInit<<<grid, block>>>(window_width, window_height, d_rand_state);
    hipDeviceSynchronize();
}

// extern "C"
// void LaunchCreateWorld(Sphere** d_list, Sphere** d_world, hiprandState* d_rand_state2)
// {
//     CreateWorld << < 1, 1 >> > (d_list, d_world, d_rand_state2);
//     hipDeviceSynchronize();
// }

// extern "C"
// void LaunchFreeWorld(Sphere** d_list, Sphere** d_world, const unsigned int num_hittables)
// {
//     FreeWorld << < 1, 1 >> > (d_list, d_world, num_hittables);
//     hipDeviceSynchronize();
// }